#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_0;
extern "C" __global__ void map(const Int64 shIn0_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const float x0 = indexArray(arrIn0_0, ix);

arrOut_0[ix] = (Int64) (1.0f == x0);
}
}
