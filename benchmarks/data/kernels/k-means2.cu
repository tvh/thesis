#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexWord32 arrIn0_2;
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
static TexWord32 arrIn1_2;
static TexFloat arrIn1_1;
static TexFloat arrIn1_0;
extern "C" __global__ void foldAll(const Int64 shIn0_0, const Int64 shIn1_0, const Int64 shOut_0, Word8* __restrict__ arrOut_0, const Int64 shRec_0, const Word8* __restrict__ arrRec_0)
{
extern volatile __shared__ Word8 sdata0[];
Word8 x0;
Word8 y0;
const Int64 sh0 = shRec_0;
const int shapeSize = sh0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;

if (ix < shapeSize) {
y0 = arrRec_0[ix];
for (ix += gridSize; ix < shapeSize; ix += gridSize) {
x0 = arrRec_0[ix];
y0 = x0 || y0;
}
}
sdata0[threadIdx.x] = y0;
ix = min(shapeSize - blockIdx.x * blockDim.x, blockDim.x);
__syncthreads();
if (threadIdx.x + 256 < ix) {
x0 = sdata0[threadIdx.x + 256];
y0 = y0 || x0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < ix) {
x0 = sdata0[threadIdx.x + 128];
y0 = y0 || x0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < ix) {
x0 = sdata0[threadIdx.x + 64];
y0 = y0 || x0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < ix) {
x0 = sdata0[threadIdx.x + 32];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < ix) {
x0 = sdata0[threadIdx.x + 16];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < ix) {
x0 = sdata0[threadIdx.x + 8];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < ix) {
x0 = sdata0[threadIdx.x + 4];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < ix) {
x0 = sdata0[threadIdx.x + 2];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < ix) {
x0 = sdata0[threadIdx.x + 1];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
if (shapeSize > 0) {
if (gridDim.x == 1) {
x0 = 0;
y0 = x0 || y0;
}
arrOut_0[blockIdx.x] = y0;
} else {
arrOut_0[blockIdx.x] = 0;
}
}
}
