#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexWord32 arrIn0_0;
extern "C" __global__ void map(const Int64 shIn0_1, const Int64 shIn0_0, const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Word32 x0 = indexArray(arrIn0_0, ix);
const Word32 v0 = (Word32) 255;

arrOut_0[ix] = 255.0f * ((0.3f * (float) (v0 & x0) + 0.59f * (float) (v0 & idiv(x0, (Word32) 256)) + 0.11f * (float) (v0 & idiv(x0, (Word32) 65536))) / 255.0f);
}
}
