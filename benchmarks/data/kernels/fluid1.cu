#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
extern "C" __global__ void generate(const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;

arrOut_0[ix] = fmaxf(0.0f, cosf(10.0f * (-50.0f + (float) sh0) / 100.0f) * cosf(10.0f * (-50.0f + (float) sh1) / 100.0f));
}
}
