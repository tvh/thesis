#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_0;
static TexFloat arrIn1_9;
static TexFloat arrIn1_8;
static TexFloat arrIn1_7;
static TexFloat arrIn1_6;
static TexFloat arrIn1_5;
static TexFloat arrIn1_4;
static TexFloat arrIn1_3;
static TexFloat arrIn1_2;
static TexFloat arrIn1_1;
static TexFloat arrIn1_0;
extern "C" __global__ void map(const Int64 shIn1_0, const Int64 shOut_0, float* __restrict__ arrOut_9, float* __restrict__ arrOut_8, float* __restrict__ arrOut_7, float* __restrict__ arrOut_6, float* __restrict__ arrOut_5, float* __restrict__ arrOut_4, float* __restrict__ arrOut_3, float* __restrict__ arrOut_2, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const float x9 = indexArray(arrIn1_9, ix);
const float x8 = indexArray(arrIn1_8, ix);
const float x7 = indexArray(arrIn1_7, ix);
const float x6 = indexArray(arrIn1_6, ix);
const float x5 = indexArray(arrIn1_5, ix);
const float x4 = indexArray(arrIn1_4, ix);
const float x3 = indexArray(arrIn1_3, ix);
float lv00 = 0.0f;
float lv01 = 0.0f;
float lv02 = 0.0f;
Int64 lv03 = (Int64) 0;
Word8 lv10;

lv10 = lv03 < shIn1_0;
while (lv10) {
Int64 lv23;
float lv22;
float lv21;
float lv20;
const float v3 = indexArray(arrIn1_9, lv03);
const float v4 = indexArray(arrIn1_8, lv03);
const float v5 = indexArray(arrIn1_7, lv03);
const float v6 = indexArray(arrIn1_6, lv03);
const float v7 = v3 - x9;
const float v8 = v4 - x8;
const float v9 = v5 - x7;
const float v10 = 1.0f / sqrtf(2500.0f + (v7 * v7 + v8 * v8 + v9 * v9));
const float v11 = v6 * (v10 * v10 * v10);
const float v12 = v11 * v7;
const float v13 = v11 * v8;
const float v14 = v11 * v9;

lv23 = (Int64) 1 + lv03;
lv22 = lv02 + v12;
lv21 = lv01 + v13;
lv20 = lv00 + v14;
lv03 = lv23;
lv02 = lv22;
lv01 = lv21;
lv00 = lv20;
lv10 = lv03 < shIn1_0;
}

const Int64 v15 = 0;
const float v16 = indexArray(arrIn0_0, v15);
const float v17 = x6 * lv02;
const float v18 = x6 * lv01;
const float v19 = x6 * lv00;
const float v20 = v16 * x5;
const float v21 = v16 * x4;
const float v22 = v16 * x3;
const float v23 = v16 * v17;
const float v24 = v16 * v18;
const float v25 = v16 * v19;

arrOut_9[ix] = x9 + v20;
arrOut_8[ix] = x8 + v21;
arrOut_7[ix] = x7 + v22;
arrOut_6[ix] = x6;
arrOut_5[ix] = x5 + v23;
arrOut_4[ix] = x4 + v24;
arrOut_3[ix] = x3 + v25;
arrOut_2[ix] = v17;
arrOut_1[ix] = v18;
arrOut_0[ix] = v19;
}
}

