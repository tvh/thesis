#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_2;
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
extern "C" __global__ void map(const Int64 shIn0_0, const Int64 shOut_0, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const float x2 = indexArray(arrIn0_2, ix);
const float x1 = indexArray(arrIn0_1, ix);
const float x0 = indexArray(arrIn0_0, ix);
const float v0 = 0.3f * sqrtf(x0);
const float v1 = x1 * expf(-(2.0e-2f * x0));
const float v2 = (logf(x2 / x1) + 6.5e-2f * x0) / v0;
const float v3 = 1.0f / (1.0f + 0.2316419f * fabsf(v2));
const float v4 = 0.3989423f * expf(-(0.5f * v2 * v2)) * (v3 * (0.31938154f + v3 * (-0.35656378f + v3 * (1.7814779f + v3 * (-1.8212559f + 1.3302745f * v3)))));
const Word8 v5 = v2 > 0.0f;
const float v6 = v5 ? 1.0f - v4 : v4;
const float v7 = v2 - v0;
const float v8 = 1.0f / (1.0f + 0.2316419f * fabsf(v7));
const float v9 = 0.3989423f * expf(-(0.5f * v7 * v7)) * (v8 * (0.31938154f + v8 * (-0.35656378f + v8 * (1.7814779f + v8 * (-1.8212559f + 1.3302745f * v8)))));
const Word8 v10 = v7 > 0.0f;
const float v11 = v10 ? 1.0f - v9 : v9;

arrOut_1[ix] = x2 * v6 - v1 * v11;
arrOut_0[ix] = v1 * (1.0f - v11) - x2 * (1.0f - v6);
}
}
