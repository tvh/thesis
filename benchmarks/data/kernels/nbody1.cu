#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_2;
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
static TexFloat arrIn1_0;
extern "C" __global__ void generate(const Int64 shIn0_0, const Int64 shIn1_0, const Int64 shOut_0, float* __restrict__ arrOut_9, float* __restrict__ arrOut_8, float* __restrict__ arrOut_7, float* __restrict__ arrOut_6, float* __restrict__ arrOut_5, float* __restrict__ arrOut_4, float* __restrict__ arrOut_3, float* __restrict__ arrOut_2, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh0 = ({ assert(ix >= 0 && ix < shOut_0); ix; });
const float v0 = indexArray(arrIn0_2, sh0);
const float v1 = indexArray(arrIn0_1, sh0);
const float v2 = indexArray(arrIn0_0, sh0);
const float v3 = indexArray(arrIn1_0, sh0);
const float v4 = sqrtf(sqrtf(v0 * v0 + v1 * v1 + v2 * v2));
const float v5 = 1.0f / sqrtf(v0 * v0 + v1 * v1 + v2 * v2);
const float v6 = v5 * v0;
const float v7 = v5 * v1;
const float v8 = v5 * v2;
const float v9 = -v6;

arrOut_9[ix] = v0;
arrOut_8[ix] = v1;
arrOut_7[ix] = v2;
arrOut_6[ix] = v3;
arrOut_5[ix] = v4 * v7;
arrOut_4[ix] = v4 * v9;
arrOut_3[ix] = v4 * v8;
arrOut_2[ix] = 0.0f;
arrOut_1[ix] = 0.0f;
arrOut_0[ix] = 0.0f;
}
}
