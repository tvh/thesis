#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexInt64 arrIn0_0;
extern "C" __global__ void scanr(const Int64 shIn0_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0, const Int64 shBlk_0, Int64* __restrict__ arrBlk_0, Int64* __restrict__ arrSum_0)
{
extern volatile __shared__ Int64 sdata0[];
Int64 x0;
Int64 y0;
Int64 z0;
const Int64 sh0 = shBlk_0;
const int shapeSize = sh0;
const int intervalSize = (shapeSize + gridDim.x - 1) / gridDim.x;
int carryIn = 0;

if (threadIdx.x == 0) {
if (gridDim.x > 1) {
z0 = arrBlk_0[blockIdx.x];
} else {
z0 = (Int64) 0;
}
}

const int start = blockIdx.x * intervalSize;
const int end = min(start + intervalSize, shapeSize);
const int numElements = end - start;
int seg;

for (seg = threadIdx.x; seg < numElements; seg += blockDim.x) {
const int ix = end - seg - 1;

x0 = arrBlk_0[ix];
if (threadIdx.x == 0) {
x0 = z0 + x0;
}
sdata0[threadIdx.x] = x0;
__syncthreads();
if (blockDim.x > 1) {
if (threadIdx.x >= 1) {
y0 = sdata0[threadIdx.x - 1];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 2) {
if (threadIdx.x >= 2) {
y0 = sdata0[threadIdx.x - 2];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 4) {
if (threadIdx.x >= 4) {
y0 = sdata0[threadIdx.x - 4];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 8) {
if (threadIdx.x >= 8) {
y0 = sdata0[threadIdx.x - 8];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 16) {
if (threadIdx.x >= 16) {
y0 = sdata0[threadIdx.x - 16];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 32) {
if (threadIdx.x >= 32) {
y0 = sdata0[threadIdx.x - 32];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 64) {
if (threadIdx.x >= 64) {
y0 = sdata0[threadIdx.x - 64];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 128) {
if (threadIdx.x >= 128) {
y0 = sdata0[threadIdx.x - 128];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 256) {
if (threadIdx.x >= 256) {
y0 = sdata0[threadIdx.x - 256];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 512) {
if (threadIdx.x >= 512) {
y0 = sdata0[threadIdx.x - 512];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (1) {
if (threadIdx.x == 0) {
x0 = z0;
} else {
x0 = sdata0[threadIdx.x - 1];
}
}
arrOut_0[ix] = x0;
if (threadIdx.x == 0) {
const int last = min(numElements - seg, blockDim.x) - 1;

z0 = sdata0[last];
}
}
if (threadIdx.x == 0 && blockIdx.x == 0) {
arrSum_0[0] = z0;
}
}
