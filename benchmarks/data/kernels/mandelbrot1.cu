#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_3;
static TexFloat arrIn0_2;
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
extern "C" __global__ void generate(const Int64 shOut_1, const Int64 shOut_0, Word32* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int32 v0 = (Int32) 255;
const Int64 v1 = 0;
const float v2 = indexArray(arrIn0_3, v1);
const float v3 = indexArray(arrIn0_2, v1);
const float v4 = indexArray(arrIn0_1, v1);
const float v5 = indexArray(arrIn0_0, v1);
const float v6 = v2 + (float) sh0 * (v4 - v2) / 1600.0f;
const float v7 = v3 + (float) sh1 * (v5 - v3) / 1200.0f;
Int32 lv80 = (Int32) 0;
float lv81 = v7;
float lv82 = v6;
Word8 lv90;

lv90 = lv80 < v0 && lv82 * lv82 + lv81 * lv81 < 4.0f;
while (lv90) {
float lv102;
float lv101;
Int32 lv100;
const float v11 = lv82 * lv82 - lv81 * lv81;
const float v12 = lv82 * lv81 + lv81 * lv82;

lv102 = v6 + v11;
lv101 = v7 + v12;
lv100 = (Int32) 1 + lv80;
lv82 = lv102;
lv81 = lv101;
lv80 = lv100;
lv90 = lv80 < v0 && lv82 * lv82 + lv81 * lv81 < 4.0f;
}

const Word8 v13 = v0 == lv80;
const Int32 v14 = v0 - lv80;
const Word8 v15 = (Word8) 0;
const Word8 v16 = (Word8) ((Int32) 7 * v14);
const Word8 v17 = (Word8) ((Int32) 5 * v14);
const Word8 v18 = (Word8) ((Int32) 3 * v14);

arrOut_0[ix] = v13 ? (Word32) 4278190080 : (Word32) 4294967295 - ((Word32) v15 + (Word32) 256 * (Word32) v16 + (Word32) 65536 * (Word32) v17 + (Word32) 16777216 * (Word32) v18);
}
}
