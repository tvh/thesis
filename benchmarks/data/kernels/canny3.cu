#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrStencil_0;
extern "C" __global__ void stencil(const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_0, const Int64 shStencil_1, const Int64 shStencil_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;

if (__all(sh1 >= 2 && sh1 < shOut_1 - 2 && (sh0 >= 1 && sh0 < shOut_0 - 1))) {
const Int64 w1 = (sh1 + -2) * shStencil_0 + (sh0 + 0);
const Int64 w4 = (sh1 + -1) * shStencil_0 + (sh0 + 0);
const Int64 w9 = (sh1 + 1) * shStencil_0 + (sh0 + 0);
const Int64 w12 = (sh1 + 2) * shStencil_0 + (sh0 + 0);
const float x13 = indexArray(arrStencil_0, w1);
const float x10 = indexArray(arrStencil_0, w4);
const float x7 = indexArray(arrStencil_0, ix);
const float x4 = indexArray(arrStencil_0, w9);
const float x1 = indexArray(arrStencil_0, w12);

arrOut_0[ix] = 3.90625e-3f * x13 + 1.5625e-2f * x10 + 2.34375e-2f * x7 + 1.5625e-2f * x4 + 3.90625e-3f * x1;
} else {
const Int64 w1 = max((Int64) 0, min(sh1 + -2, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 0, shStencil_0 - 1));
const Int64 w4 = max((Int64) 0, min(sh1 + -1, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 0, shStencil_0 - 1));
const Int64 w9 = max((Int64) 0, min(sh1 + 1, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 0, shStencil_0 - 1));
const Int64 w12 = max((Int64) 0, min(sh1 + 2, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 0, shStencil_0 - 1));
const float x13 = indexArray(arrStencil_0, w1);
const float x10 = indexArray(arrStencil_0, w4);
const float x7 = indexArray(arrStencil_0, ix);
const float x4 = indexArray(arrStencil_0, w9);
const float x1 = indexArray(arrStencil_0, w12);

arrOut_0[ix] = 3.90625e-3f * x13 + 1.5625e-2f * x10 + 2.34375e-2f * x7 + 1.5625e-2f * x4 + 3.90625e-3f * x1;
}
}
}
