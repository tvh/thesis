#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_1;
static TexInt64 arrIn0_0;
extern "C" __global__ void generate(const Int64 shIn0_1, const Int64 shIn0_0, const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int64 v0 = (Int64) 1;
const Int64 v1 = sh1 * shIn0_0 + sh0;
const float v2 = indexArray(arrIn0_1, v1);
const Int64 v3 = indexArray(arrIn0_0, v1);
const Int64 v4 = (Int64) 0;
const Int64 v5 = (Int64) -1;
const Int64 v6 = (Int64) -1 + shIn0_0;
const Int64 v7 = (Int64) -1 + shIn0_1;
const Word8 v8 = v3 < (Int64) 255;
const Int64 v9 = v8 ? v5 : v4;
const Int64 v10 = (Int64) 128;
const Word8 v11 = v3 > v10;
const Word8 v12 = v3 < v10;
const Int64 v13 = v11 ? v5 : v12 ? v0 : v4;
const Int64 v14 = min(max(v4, sh1 + v9), v7) * shIn0_0 + min(max(v4, sh0 + v13), v6);
const Int64 v15 = min(max(v4, sh1 - v9), v7) * shIn0_0 + min(max(v4, sh0 - v13), v6);

arrOut_0[ix] = 0.5f * (float) ((Int64) !(v4 == v3 || (v2 < 50.0f || (v2 < indexArray(arrIn0_1, v14) || v2 < indexArray(arrIn0_1, v15)))) * (v0 + (Int64) (v2 >= 100.0f)));
}
}
