#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexInt64 arrIn0_0;
extern "C" __global__ void generate(const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
arrOut_0[ix] = (Int64) 0;
}
}
