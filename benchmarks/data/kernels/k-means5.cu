#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
static TexWord32 arrIn1_2;
static TexFloat arrIn1_1;
static TexFloat arrIn1_0;
extern "C" __global__ void fold1(const Int64 shIn0_0, const Int64 shIn1_0, const Int64 shOut_0, Word32* __restrict__ arrOut_2, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0)
{
extern volatile __shared__ Word32 sdata2[];
volatile float* sdata1 = (float*) &sdata2[blockDim.x];
volatile float* sdata0 = (float*) &sdata1[blockDim.x];
Word32 x2;
float x1;
float x0;
Word32 y2;
float y1;
float y0;
const Int64 sh1 = shIn1_0;
const Int64 sh0 = shIn0_0;
const int numIntervals = sh1;
const int intervalSize = sh0;
int ix;
int seg;

for (seg = blockIdx.x; seg < numIntervals; seg += gridDim.x) {
const int start = seg * intervalSize;
const int end = start + intervalSize;
const int n = min(end - start, blockDim.x);

if (threadIdx.x >= n)
return;

ix = start - (start & warpSize - 1);
if (ix == start || intervalSize > blockDim.x) {
ix += threadIdx.x;
if (ix >= start) {
const Int64 v13_0 = ix;
const Int64 v13_1 = v13_0 / shIn0_0;
const Int64 v14 = v13_1 % shIn1_0;
const Int64 v15 = v13_0 % shIn0_0;
const float v16 = indexArray(arrIn0_1, v15);
const float v17 = indexArray(arrIn0_0, v15);
float lv180 = 3.4028235e38f;
Word32 lv181 = (Word32) 4294967295;
Int64 lv182 = (Int64) 0;
Word8 lv190;

lv190 = lv182 < shIn1_0;
while (lv190) {
Int64 lv202;
Word32 lv201;
float lv200;
const Word32 v21 = indexArray(arrIn1_2, lv182);
const float v22 = indexArray(arrIn1_1, lv182);
const float v23 = indexArray(arrIn1_0, lv182);
const float v24 = v16 - v22;
const float v25 = v17 - v23;
const float v26 = v24 * v24 + v25 * v25;
const Word8 v27 = v26 < lv180;

lv202 = (Int64) 1 + lv182;
lv201 = v27 ? v21 : lv181;
lv200 = v27 ? v26 : lv180;
lv182 = lv202;
lv181 = lv201;
lv180 = lv200;
lv190 = lv182 < shIn1_0;
}

const Word8 v28 = lv181 == (Word32) v14;

y2 = v28 ? (Word32) 1 : (Word32) 0;
y1 = v28 ? indexArray(arrIn0_1, v15) : 0.0f;
y0 = v28 ? indexArray(arrIn0_0, v15) : 0.0f;
}
if (ix + blockDim.x < end) {
const Int64 v13_0 = ix + blockDim.x;
const Int64 v13_1 = v13_0 / shIn0_0;
const Int64 v14 = v13_1 % shIn1_0;
const Int64 v15 = v13_0 % shIn0_0;
const float v16 = indexArray(arrIn0_1, v15);
const float v17 = indexArray(arrIn0_0, v15);
float lv180 = 3.4028235e38f;
Word32 lv181 = (Word32) 4294967295;
Int64 lv182 = (Int64) 0;
Word8 lv190;

lv190 = lv182 < shIn1_0;
while (lv190) {
Int64 lv202;
Word32 lv201;
float lv200;
const Word32 v21 = indexArray(arrIn1_2, lv182);
const float v22 = indexArray(arrIn1_1, lv182);
const float v23 = indexArray(arrIn1_0, lv182);
const float v24 = v16 - v22;
const float v25 = v17 - v23;
const float v26 = v24 * v24 + v25 * v25;
const Word8 v27 = v26 < lv180;

lv202 = (Int64) 1 + lv182;
lv201 = v27 ? v21 : lv181;
lv200 = v27 ? v26 : lv180;
lv182 = lv202;
lv181 = lv201;
lv180 = lv200;
lv190 = lv182 < shIn1_0;
}

const Word8 v28 = lv181 == (Word32) v14;

x2 = v28 ? (Word32) 1 : (Word32) 0;
x1 = v28 ? indexArray(arrIn0_1, v15) : 0.0f;
x0 = v28 ? indexArray(arrIn0_0, v15) : 0.0f;
if (ix >= start) {
y2 = x2 + y2;
y1 = x1 + y1;
y0 = x0 + y0;
} else {
y2 = x2;
y1 = x1;
y0 = x0;
}
}
for (ix += 2 * blockDim.x; ix < end; ix += blockDim.x) {
const Int64 v13_0 = ix;
const Int64 v13_1 = v13_0 / shIn0_0;
const Int64 v14 = v13_1 % shIn1_0;
const Int64 v15 = v13_0 % shIn0_0;
const float v16 = indexArray(arrIn0_1, v15);
const float v17 = indexArray(arrIn0_0, v15);
float lv180 = 3.4028235e38f;
Word32 lv181 = (Word32) 4294967295;
Int64 lv182 = (Int64) 0;
Word8 lv190;

lv190 = lv182 < shIn1_0;
while (lv190) {
Int64 lv202;
Word32 lv201;
float lv200;
const Word32 v21 = indexArray(arrIn1_2, lv182);
const float v22 = indexArray(arrIn1_1, lv182);
const float v23 = indexArray(arrIn1_0, lv182);
const float v24 = v16 - v22;
const float v25 = v17 - v23;
const float v26 = v24 * v24 + v25 * v25;
const Word8 v27 = v26 < lv180;

lv202 = (Int64) 1 + lv182;
lv201 = v27 ? v21 : lv181;
lv200 = v27 ? v26 : lv180;
lv182 = lv202;
lv181 = lv201;
lv180 = lv200;
lv190 = lv182 < shIn1_0;
}

const Word8 v28 = lv181 == (Word32) v14;

x2 = v28 ? (Word32) 1 : (Word32) 0;
x1 = v28 ? indexArray(arrIn0_1, v15) : 0.0f;
x0 = v28 ? indexArray(arrIn0_0, v15) : 0.0f;
y2 = x2 + y2;
y1 = x1 + y1;
y0 = x0 + y0;
}
} else {
const Int64 v13_0 = start + threadIdx.x;
const Int64 v13_1 = v13_0 / shIn0_0;
const Int64 v14 = v13_1 % shIn1_0;
const Int64 v15 = v13_0 % shIn0_0;
const float v16 = indexArray(arrIn0_1, v15);
const float v17 = indexArray(arrIn0_0, v15);
float lv180 = 3.4028235e38f;
Word32 lv181 = (Word32) 4294967295;
Int64 lv182 = (Int64) 0;
Word8 lv190;

lv190 = lv182 < shIn1_0;
while (lv190) {
Int64 lv202;
Word32 lv201;
float lv200;
const Word32 v21 = indexArray(arrIn1_2, lv182);
const float v22 = indexArray(arrIn1_1, lv182);
const float v23 = indexArray(arrIn1_0, lv182);
const float v24 = v16 - v22;
const float v25 = v17 - v23;
const float v26 = v24 * v24 + v25 * v25;
const Word8 v27 = v26 < lv180;

lv202 = (Int64) 1 + lv182;
lv201 = v27 ? v21 : lv181;
lv200 = v27 ? v26 : lv180;
lv182 = lv202;
lv181 = lv201;
lv180 = lv200;
lv190 = lv182 < shIn1_0;
}

const Word8 v28 = lv181 == (Word32) v14;

y2 = v28 ? (Word32) 1 : (Word32) 0;
y1 = v28 ? indexArray(arrIn0_1, v15) : 0.0f;
y0 = v28 ? indexArray(arrIn0_0, v15) : 0.0f;
}
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 256 < n) {
x2 = sdata2[threadIdx.x + 256];
x1 = sdata1[threadIdx.x + 256];
x0 = sdata0[threadIdx.x + 256];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
}
__syncthreads();
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < n) {
x2 = sdata2[threadIdx.x + 128];
x1 = sdata1[threadIdx.x + 128];
x0 = sdata0[threadIdx.x + 128];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
}
__syncthreads();
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < n) {
x2 = sdata2[threadIdx.x + 64];
x1 = sdata1[threadIdx.x + 64];
x0 = sdata0[threadIdx.x + 64];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
}
__syncthreads();
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < n) {
x2 = sdata2[threadIdx.x + 32];
x1 = sdata1[threadIdx.x + 32];
x0 = sdata0[threadIdx.x + 32];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < n) {
x2 = sdata2[threadIdx.x + 16];
x1 = sdata1[threadIdx.x + 16];
x0 = sdata0[threadIdx.x + 16];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < n) {
x2 = sdata2[threadIdx.x + 8];
x1 = sdata1[threadIdx.x + 8];
x0 = sdata0[threadIdx.x + 8];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < n) {
x2 = sdata2[threadIdx.x + 4];
x1 = sdata1[threadIdx.x + 4];
x0 = sdata0[threadIdx.x + 4];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < n) {
x2 = sdata2[threadIdx.x + 2];
x1 = sdata1[threadIdx.x + 2];
x0 = sdata0[threadIdx.x + 2];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < n) {
x2 = sdata2[threadIdx.x + 1];
x1 = sdata1[threadIdx.x + 1];
x0 = sdata0[threadIdx.x + 1];
y2 = y2 + x2;
y1 = y1 + x1;
y0 = y0 + x0;
sdata2[threadIdx.x] = y2;
sdata1[threadIdx.x] = y1;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
arrOut_2[seg] = y2;
arrOut_1[seg] = y1;
arrOut_0[seg] = y0;
}
}
}
