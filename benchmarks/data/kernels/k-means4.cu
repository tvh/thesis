#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexWord32 arrIn0_2;
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
extern "C" __global__ void generate(const Int64 shIn0_0, const Int64 shOut_0, Word32* __restrict__ arrOut_2, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh0 = ({ assert(ix >= 0 && ix < shOut_0); ix; });
const Word32 v0 = indexArray(arrIn0_2, sh0);
const float v1 = indexArray(arrIn0_1, sh0);
const float v2 = indexArray(arrIn0_0, sh0);

arrOut_2[ix] = (Word32) sh0;
arrOut_1[ix] = v1 / (float) v0;
arrOut_0[ix] = v2 / (float) v0;
}
}
