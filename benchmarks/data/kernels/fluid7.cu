#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrStencil1_1;
static TexFloat arrStencil1_0;
static TexFloat arrStencil2_0;
extern "C" __global__ void stencil2(const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0, const Int64 shStencil1_1, const Int64 shStencil1_0, const Int64 shStencil2_1, const Int64 shStencil2_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;

if (__all(sh1 >= 1 && sh1 < shOut_1 - 1 && (sh0 >= 1 && sh0 < shOut_0 - 1))) {
const Int64 w4 = sh1 * shStencil1_0 + sh0;
const float x9 = indexArray(arrStencil1_1, w4);
const float x8 = indexArray(arrStencil1_0, w4);
const Int64 z1 = (sh1 + -1) * shStencil2_0 + (sh0 + 0);
const Int64 z3 = (sh1 + 0) * shStencil2_0 + (sh0 + -1);
const Int64 z5 = (sh1 + 0) * shStencil2_0 + (sh0 + 1);
const Int64 z7 = (sh1 + 1) * shStencil2_0 + (sh0 + 0);
const float y7 = indexArray(arrStencil2_0, z1);
const float y5 = indexArray(arrStencil2_0, z3);
const float y3 = indexArray(arrStencil2_0, z5);
const float y1 = indexArray(arrStencil2_0, z7);
const float v0 = y3 - y5;
const float v1 = y1 - y7;
const float v2 = 0.5f * v0;
const float v3 = 0.5f * v1;

arrOut_1[ix] = x9 - v2;
arrOut_0[ix] = x8 - v3;
} else {
const Int64 w8 = sh1 * shStencil1_0 + sh0;
const float x9 = indexArray(arrStencil1_1, w8);
const float x8 = indexArray(arrStencil1_0, w8);
const int z3 = ({ const Int64 _i = sh1 + -1;  _i >= 0 && _i < shStencil2_1; }) && ({ const Int64 _i = sh0 + 0;  _i >= 0 && _i < shStencil2_0; });
const Int64 z2 = (sh1 + -1) * shStencil2_0 + (sh0 + 0);
const int z7 = ({ const Int64 _i = sh1 + 0;  _i >= 0 && _i < shStencil2_1; }) && ({ const Int64 _i = sh0 + -1;  _i >= 0 && _i < shStencil2_0; });
const Int64 z6 = (sh1 + 0) * shStencil2_0 + (sh0 + -1);
const int z10 = ({ const Int64 _i = sh1 + 0;  _i >= 0 && _i < shStencil2_1; }) && ({ const Int64 _i = sh0 + 1;  _i >= 0 && _i < shStencil2_0; });
const Int64 z9 = (sh1 + 0) * shStencil2_0 + (sh0 + 1);
const int z14 = ({ const Int64 _i = sh1 + 1;  _i >= 0 && _i < shStencil2_1; }) && ({ const Int64 _i = sh0 + 0;  _i >= 0 && _i < shStencil2_0; });
const Int64 z13 = (sh1 + 1) * shStencil2_0 + (sh0 + 0);
const float y7 = z3 ? indexArray(arrStencil2_0, z2) : 0.0f;
const float y5 = z7 ? indexArray(arrStencil2_0, z6) : 0.0f;
const float y3 = z10 ? indexArray(arrStencil2_0, z9) : 0.0f;
const float y1 = z14 ? indexArray(arrStencil2_0, z13) : 0.0f;
const float v0 = y3 - y5;
const float v1 = y1 - y7;
const float v2 = 0.5f * v0;
const float v3 = 0.5f * v1;

arrOut_1[ix] = x9 - v2;
arrOut_0[ix] = x8 - v3;
}
}
}
