#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
extern "C" __global__ void generate(const Int64 shIn0_1, const Int64 shIn0_0, const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int64 v0 = (Int64) 0;
const Int64 v1 = (Int64) 1;
const Int64 v2 = sh1 * shIn0_0 + sh0;
const float v3 = indexArray(arrIn0_1, v2);
const float v4 = indexArray(arrIn0_0, v2);
const float v5 = (float) shIn0_0;
const float v6 = fmaxf(-0.5f, fminf(0.5f + v5, (float) sh0 - 0.1f * v5 * v3));
const float v7 = (float) shIn0_1;
const float v8 = fmaxf(-0.5f, fminf(0.5f + v7, (float) sh1 - 0.1f * v7 * v4));
const Int64 v9 = (Int64) -1 + (Int64) truncf(1.0f + v6);
const Int64 v10 = (Int64) -1 + (Int64) truncf(1.0f + v8);
const float v11 = v6 - (float) v9;
const float v12 = v8 - (float) v10;
const Int64 v13 = v1 + v10;
const float v14 = 1.0f - v12;
const float v15 = 1.0f - v11;
const Word8 v16 = v10 < v0 || (v9 < v0 || (v10 >= shIn0_1 || v9 >= shIn0_0));
const Int64 v17 = v10 * shIn0_0 + v9;
const float v18 = v16 ? 0.0f : indexArray(arrIn0_1, v17);
const float v19 = v16 ? 0.0f : indexArray(arrIn0_0, v17);
const float v20 = v14 * v18;
const float v21 = v14 * v19;
const Word8 v22 = v13 < v0 || (v9 < v0 || (v13 >= shIn0_1 || v9 >= shIn0_0));
const Int64 v23 = v13 * shIn0_0 + v9;
const float v24 = v22 ? 0.0f : indexArray(arrIn0_1, v23);
const float v25 = v22 ? 0.0f : indexArray(arrIn0_0, v23);
const float v26 = v12 * v24;
const float v27 = v12 * v25;
const float v28 = v20 + v26;
const float v29 = v21 + v27;
const float v30 = v15 * v28;
const float v31 = v15 * v29;
const Int64 v32 = v1 + v9;
const Word8 v33 = v10 < v0 || (v32 < v0 || (v10 >= shIn0_1 || v32 >= shIn0_0));
const Int64 v34 = v10 * shIn0_0 + v32;
const float v35 = v33 ? 0.0f : indexArray(arrIn0_1, v34);
const float v36 = v33 ? 0.0f : indexArray(arrIn0_0, v34);
const float v37 = v14 * v35;
const float v38 = v14 * v36;
const Word8 v39 = v13 < v0 || (v32 < v0 || (v13 >= shIn0_1 || v32 >= shIn0_0));
const Int64 v40 = v13 * shIn0_0 + v32;
const float v41 = v39 ? 0.0f : indexArray(arrIn0_1, v40);
const float v42 = v39 ? 0.0f : indexArray(arrIn0_0, v40);
const float v43 = v12 * v41;
const float v44 = v12 * v42;
const float v45 = v37 + v43;
const float v46 = v38 + v44;
const float v47 = v11 * v45;
const float v48 = v11 * v46;

arrOut_1[ix] = v30 + v47;
arrOut_0[ix] = v31 + v48;
}
}
