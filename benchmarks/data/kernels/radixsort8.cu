#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexInt64 arrIn0_0;
static TexInt64 arrIn1_0;
static TexInt64 arrIn2_0;
static TexInt32 arrIn3_0;
extern "C" __global__ void permute(const Int64 shIn0_0, const Int64 shIn1_0, const Int64 shIn2_0, const Int64 shIn3_0, const Int64 shOut_0, Int32* __restrict__ arrOut_0)
{
const Int64 shIn0 = shIn3_0;
const int shapeSize = shIn0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh0 = ({ assert(ix >= 0 && ix < shIn0); ix; });
const Int64 v0 = indexArray(arrIn1_0, sh0);
const Int64 v1 = (Int64) -1 + shIn3_0 - indexArray(arrIn0_0, sh0);
const Word8 v2 = (Int64) 0 == indexArray(arrIn2_0, sh0);
const Int64 sh_0 = v2 ? v0 : v1;

if (!(sh_0 == -1)) {
Int32 y0;
Int32 _y0;
const Int64 jx0 = sh_0;
const Int32 x0 = indexArray(arrIn3_0, ix);

arrOut_0[jx0] = x0;
}
}
}
