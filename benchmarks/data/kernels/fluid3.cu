#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexInt64 arrIn0_1;
static TexInt64 arrIn0_0;
static TexFloat arrIn1_1;
static TexFloat arrIn1_0;
extern "C" __global__ void permute(const Int64 shIn0_0, const Int64 shIn1_0, const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0)
{
const Int64 shIn0 = shIn1_0;
const int shapeSize = shIn0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh0 = ({ assert(ix >= 0 && ix < shIn0); ix; });
const Int64 sh_1 = indexArray(arrIn0_1, sh0);
const Int64 sh_0 = indexArray(arrIn0_0, sh0);

if (!(sh_1 == -1 && sh_0 == -1)) {
float y1;
float y0;
float _y1;
float _y0;
const Int64 jx0 = sh_1 * shOut_0 + sh_0;
const float x1 = indexArray(arrIn1_1, ix);
const float x0 = indexArray(arrIn1_0, ix);

y1 = arrOut_1[jx0];
y0 = arrOut_0[jx0];
do {
_y1 = y1;
y1 = atomicCAS32(&arrOut_1[jx0], _y1, x1 + y1);
} while(y1 != _y1);
do {
_y0 = y0;
y0 = atomicCAS32(&arrOut_0[jx0], _y0, x0 + y0);
} while(y0 != _y0);
}
}
}
