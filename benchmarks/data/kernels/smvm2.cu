#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexInt32 arrIn0_0;
extern "C" __global__ void scanlUp(const Int64 shIn0_0, const Int64 shOut_0, Int32* __restrict__ arrOut_0)
{
extern volatile __shared__ Int32 sdata0[];
Int32 x0;
Int32 y0;
const Int64 sh0 = shIn0_0;
const int shapeSize = sh0;
const int intervalSize = (shapeSize + gridDim.x - 1) / gridDim.x;
const int start = blockIdx.x * intervalSize;
const int end = min(start + intervalSize, shapeSize);
const int numElements = end - start;
int carryIn = 0;
int seg;

for (seg = threadIdx.x; seg < numElements; seg += blockDim.x) {
const int ix = start + seg;

x0 = indexArray(arrIn0_0, ix);
if (threadIdx.x == 0 && carryIn) {
x0 = y0 + x0;
}
sdata0[threadIdx.x] = x0;
__syncthreads();
if (blockDim.x > 1) {
if (threadIdx.x >= 1) {
y0 = sdata0[threadIdx.x - 1];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 2) {
if (threadIdx.x >= 2) {
y0 = sdata0[threadIdx.x - 2];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 4) {
if (threadIdx.x >= 4) {
y0 = sdata0[threadIdx.x - 4];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 8) {
if (threadIdx.x >= 8) {
y0 = sdata0[threadIdx.x - 8];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 16) {
if (threadIdx.x >= 16) {
y0 = sdata0[threadIdx.x - 16];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 32) {
if (threadIdx.x >= 32) {
y0 = sdata0[threadIdx.x - 32];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 64) {
if (threadIdx.x >= 64) {
y0 = sdata0[threadIdx.x - 64];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 128) {
if (threadIdx.x >= 128) {
y0 = sdata0[threadIdx.x - 128];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 256) {
if (threadIdx.x >= 256) {
y0 = sdata0[threadIdx.x - 256];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (blockDim.x > 512) {
if (threadIdx.x >= 512) {
y0 = sdata0[threadIdx.x - 512];
x0 = y0 + x0;
}
__syncthreads();
sdata0[threadIdx.x] = x0;
__syncthreads();
}
if (threadIdx.x == 0) {
const int last = min(numElements - seg, blockDim.x) - 1;

y0 = sdata0[last];
}
carryIn = 1;
}
if (threadIdx.x == 0) {
arrOut_0[blockIdx.x] = y0;
}
}
