#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexInt64 arrIn0_0;
static TexInt32 arrIn1_0;
extern "C" __global__ void generate(const Int64 shIn1_1, const Int64 shIn1_0, const Int64 shOut_1, const Int64 shOut_0, Int32* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int64 v0 = sh1 * shIn1_0 + sh0;
const Int64 v1 = 0;
const Int64 v2 = indexArray(arrIn0_0, v1);
const Int64 v3 = sh1 * shIn1_0 + v2;
const Int64 v4 = v2 * shIn1_0 + sh0;

arrOut_0[ix] = min(indexArray(arrIn1_0, v0), indexArray(arrIn1_0, v3) + indexArray(arrIn1_0, v4));
}
}
