#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrStencil1_1;
static TexFloat arrStencil1_0;
static TexFloat arrStencil2_0;
extern "C" __global__ void stencil2(const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_1, float* __restrict__ arrOut_0, const Int64 shStencil1_1, const Int64 shStencil1_0, const Int64 shStencil2_1, const Int64 shStencil2_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;

if (__all(sh1 >= 1 && sh1 < shOut_1 - 1 && (sh0 >= 1 && sh0 < shOut_0 - 1))) {
const float x9 = indexArray(arrStencil1_1, ix);
const float x8 = indexArray(arrStencil1_0, ix);
const Int64 z1 = (sh1 + -1) * shStencil1_0 + (sh0 + 0);
const Int64 z3 = (sh1 + 0) * shStencil1_0 + (sh0 + -1);
const Int64 z4 = (sh1 + 0) * shStencil1_0 + (sh0 + 1);
const Int64 z6 = (sh1 + 1) * shStencil1_0 + (sh0 + 0);
const float y7 = indexArray(arrStencil2_0, z1);
const float y5 = indexArray(arrStencil2_0, z3);
const float y3 = indexArray(arrStencil2_0, z4);
const float y1 = indexArray(arrStencil2_0, z6);
const float v0 = y3 - y5;
const float v1 = y1 - y7;
const float v2 = 0.5f * v0;
const float v3 = 0.5f * v1;

arrOut_1[ix] = x9 - v2;
arrOut_0[ix] = x8 - v3;
} else {
const float x9 = indexArray(arrStencil1_1, ix);
const float x8 = indexArray(arrStencil1_0, ix);
const int z3 = ({ const Int64 _i = sh1 + -1;  _i >= 0 && _i < shStencil1_1; }) && ({ const Int64 _i = sh0 + 0;  _i >= 0 && _i < shStencil1_0; });
const Int64 z2 = (sh1 + -1) * shStencil1_0 + (sh0 + 0);
const int z7 = ({ const Int64 _i = sh1 + 0;  _i >= 0 && _i < shStencil1_1; }) && ({ const Int64 _i = sh0 + -1;  _i >= 0 && _i < shStencil1_0; });
const Int64 z6 = (sh1 + 0) * shStencil1_0 + (sh0 + -1);
const int z9 = ({ const Int64 _i = sh1 + 0;  _i >= 0 && _i < shStencil1_1; }) && ({ const Int64 _i = sh0 + 1;  _i >= 0 && _i < shStencil1_0; });
const Int64 z8 = (sh1 + 0) * shStencil1_0 + (sh0 + 1);
const int z13 = ({ const Int64 _i = sh1 + 1;  _i >= 0 && _i < shStencil1_1; }) && ({ const Int64 _i = sh0 + 0;  _i >= 0 && _i < shStencil1_0; });
const Int64 z12 = (sh1 + 1) * shStencil1_0 + (sh0 + 0);
const float y7 = z3 ? indexArray(arrStencil2_0, z2) : 0.0f;
const float y5 = z7 ? indexArray(arrStencil2_0, z6) : 0.0f;
const float y3 = z9 ? indexArray(arrStencil2_0, z8) : 0.0f;
const float y1 = z13 ? indexArray(arrStencil2_0, z12) : 0.0f;
const float v0 = y3 - y5;
const float v1 = y1 - y7;
const float v2 = 0.5f * v0;
const float v3 = 0.5f * v1;

arrOut_1[ix] = x9 - v2;
arrOut_0[ix] = x8 - v3;
}
}
}
