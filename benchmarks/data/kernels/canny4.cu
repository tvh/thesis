#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrStencil_0;
extern "C" __global__ void stencil(const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_1, Int64* __restrict__ arrOut_0, const Int64 shStencil_1, const Int64 shStencil_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;

if (__all(sh1 >= 1 && sh1 < shOut_1 - 1 && (sh0 >= 1 && sh0 < shOut_0 - 1))) {
const Int64 w0 = (sh1 + -1) * shStencil_0 + (sh0 + -1);
const Int64 w1 = (sh1 + -1) * shStencil_0 + (sh0 + 0);
const Int64 w2 = (sh1 + -1) * shStencil_0 + (sh0 + 1);
const Int64 w3 = (sh1 + 0) * shStencil_0 + (sh0 + -1);
const Int64 w4 = (sh1 + 0) * shStencil_0 + (sh0 + 1);
const Int64 w5 = (sh1 + 1) * shStencil_0 + (sh0 + -1);
const Int64 w6 = (sh1 + 1) * shStencil_0 + (sh0 + 0);
const Int64 w7 = (sh1 + 1) * shStencil_0 + (sh0 + 1);
const float x8 = indexArray(arrStencil_0, w0);
const float x7 = indexArray(arrStencil_0, w1);
const float x6 = indexArray(arrStencil_0, w2);
const float x5 = indexArray(arrStencil_0, w3);
const float x3 = indexArray(arrStencil_0, w4);
const float x2 = indexArray(arrStencil_0, w5);
const float x1 = indexArray(arrStencil_0, w6);
const float x0 = indexArray(arrStencil_0, w7);
const float v0 = x6 + 2.0f * x3 + x0 - x8 - 2.0f * x5 - x2;
const float v1 = x8 + 2.0f * x7 + x6 - x2 - 2.0f * x1 - x0;
const float v2 = 1.2732395f * (-0.3926991f + atan2f(v1, v0));

arrOut_1[ix] = sqrtf(v0 * v0 + v1 * v1);
arrOut_0[ix] = (Int64) !(fabsf(v0) <= 50.0f && fabsf(v1) <= 50.0f) * min((Int64) 255, (Int64) 64 * ((Int64) 1 + mod((Int64) floorf(v2 + 8.0f * (float) (Int64) (v2 <= 0.0f)), (Int64) 4)));
} else {
const Int64 w0 = max((Int64) 0, min(sh1 + -1, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + -1, shStencil_0 - 1));
const Int64 w1 = max((Int64) 0, min(sh1 + -1, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 0, shStencil_0 - 1));
const Int64 w2 = max((Int64) 0, min(sh1 + -1, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 1, shStencil_0 - 1));
const Int64 w3 = max((Int64) 0, min(sh1 + 0, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + -1, shStencil_0 - 1));
const Int64 w4 = max((Int64) 0, min(sh1 + 0, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 1, shStencil_0 - 1));
const Int64 w5 = max((Int64) 0, min(sh1 + 1, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + -1, shStencil_0 - 1));
const Int64 w6 = max((Int64) 0, min(sh1 + 1, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 0, shStencil_0 - 1));
const Int64 w7 = max((Int64) 0, min(sh1 + 1, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 1, shStencil_0 - 1));
const float x8 = indexArray(arrStencil_0, w0);
const float x7 = indexArray(arrStencil_0, w1);
const float x6 = indexArray(arrStencil_0, w2);
const float x5 = indexArray(arrStencil_0, w3);
const float x3 = indexArray(arrStencil_0, w4);
const float x2 = indexArray(arrStencil_0, w5);
const float x1 = indexArray(arrStencil_0, w6);
const float x0 = indexArray(arrStencil_0, w7);
const float v0 = x6 + 2.0f * x3 + x0 - x8 - 2.0f * x5 - x2;
const float v1 = x8 + 2.0f * x7 + x6 - x2 - 2.0f * x1 - x0;
const float v2 = 1.2732395f * (-0.3926991f + atan2f(v1, v0));

arrOut_1[ix] = sqrtf(v0 * v0 + v1 * v1);
arrOut_0[ix] = (Int64) !(fabsf(v0) <= 50.0f && fabsf(v1) <= 50.0f) * min((Int64) 255, (Int64) 64 * ((Int64) 1 + mod((Int64) floorf(v2 + 8.0f * (float) (Int64) (v2 <= 0.0f)), (Int64) 4)));
}
}
}
