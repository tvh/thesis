#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_0;
static TexInt64 arrIn1_0;
static TexInt64 arrIn2_0;
extern "C" __global__ void permute(const Int64 shIn0_1, const Int64 shIn0_0, const Int64 shIn1_0, const Int64 shIn2_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const Int64 shIn0 = shIn0_1 * shIn0_0;
const int shapeSize = shIn0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 sh0 = ({ assert(ix >= 0 && ix < shIn0); ix; });
const Word8 v0 = (Int64) 0 == indexArray(arrIn2_0, sh0);
const Int64 sh_0 = v0 ? (Int64) -1 : indexArray(arrIn1_0, sh0);

if (!(sh_0 == -1)) {
Int64 y0;
Int64 _y0;
const Int64 jx0 = sh_0;
const Int64 v3 = ({ assert(ix >= 0 && ix < shIn0_1 * shIn0_0); ix; });
const Int64 x0 = (Int64) ({ assert(v3 >= 0 && v3 < shIn0_1 * shIn0_0); v3; });

arrOut_0[jx0] = x0;
}
}
}
