#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexInt64 arrIn0_0;
static TexInt32 arrIn1_0;
extern "C" __global__ void map(const Int64 shIn1_0, const Int64 shOut_0, Int64* __restrict__ arrOut_0)
{
const int shapeSize = shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int32 x0 = indexArray(arrIn1_0, ix);
const Int32 v0 = (Int32) 1;
const Int64 v1 = 0;
const Int64 v2 = indexArray(arrIn0_0, v1);
const Word8 v3 = (Int64) 31 == v2;

arrOut_0[ix] = v3 ? (Int64) (v0 & ((Int32) -2147483648 ^ x0) >> v2) : (Int64) (v0 & x0 >> v2);
}
}
