#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrStencil_0;
extern "C" __global__ void stencil(const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_0, const Int64 shStencil_1, const Int64 shStencil_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;

if (__all(sh1 >= 1 && sh1 < shOut_1 - 1 && (sh0 >= 2 && sh0 < shOut_0 - 2))) {
const Int64 w3 = (sh1 + 0) * shStencil_0 + (sh0 + -2);
const Int64 w4 = (sh1 + 0) * shStencil_0 + (sh0 + -1);
const Int64 w5 = (sh1 + 0) * shStencil_0 + (sh0 + 1);
const Int64 w6 = (sh1 + 0) * shStencil_0 + (sh0 + 2);
const float x7 = indexArray(arrStencil_0, w3);
const float x6 = indexArray(arrStencil_0, w4);
const float x5 = indexArray(arrStencil_0, ix);
const float x4 = indexArray(arrStencil_0, w5);
const float x3 = indexArray(arrStencil_0, w6);

arrOut_0[ix] = x7 + 4.0f * x6 + 6.0f * x5 + 4.0f * x4 + x3;
} else {
const Int64 w3 = max((Int64) 0, min(sh1 + 0, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + -2, shStencil_0 - 1));
const Int64 w4 = max((Int64) 0, min(sh1 + 0, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + -1, shStencil_0 - 1));
const Int64 w5 = max((Int64) 0, min(sh1 + 0, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 1, shStencil_0 - 1));
const Int64 w6 = max((Int64) 0, min(sh1 + 0, shStencil_1 - 1)) * shStencil_0 + max((Int64) 0, min(sh0 + 2, shStencil_0 - 1));
const float x7 = indexArray(arrStencil_0, w3);
const float x6 = indexArray(arrStencil_0, w4);
const float x5 = indexArray(arrStencil_0, ix);
const float x4 = indexArray(arrStencil_0, w5);
const float x3 = indexArray(arrStencil_0, w6);

arrOut_0[ix] = x7 + 4.0f * x6 + 6.0f * x5 + 4.0f * x4 + x3;
}
}
}
