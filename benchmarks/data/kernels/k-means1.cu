#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexWord32 arrIn0_2;
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
static TexWord32 arrIn1_2;
static TexFloat arrIn1_1;
static TexFloat arrIn1_0;
extern "C" __global__ void foldAll(const Int64 shIn0_0, const Int64 shIn1_0, const Int64 shOut_0, Word8* __restrict__ arrOut_0)
{
extern volatile __shared__ Word8 sdata0[];
Word8 x0;
Word8 y0;
const Int64 sh0 = min(shIn1_0, shIn0_0);
const int shapeSize = sh0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;

if (ix < shapeSize) {
const Int64 v7 = ({ assert(ix >= 0 && ix < min(shIn1_0, shIn0_0)); ix; });
const Int64 v9 = ({ assert(v7 >= 0 && v7 < min(shIn1_0, shIn0_0)); v7; });
const float v10 = indexArray(arrIn0_1, v9);
const float v11 = indexArray(arrIn0_0, v9);
const float v12 = indexArray(arrIn1_1, v9);
const float v13 = indexArray(arrIn1_0, v9);

y0 = fabsf(v12 - v10) > 1.0e-3f || fabsf(v13 - v11) > 1.0e-3f;
for (ix += gridSize; ix < shapeSize; ix += gridSize) {
const Int64 v7 = ({ assert(ix >= 0 && ix < min(shIn1_0, shIn0_0)); ix; });
const Int64 v9 = ({ assert(v7 >= 0 && v7 < min(shIn1_0, shIn0_0)); v7; });
const float v10 = indexArray(arrIn0_1, v9);
const float v11 = indexArray(arrIn0_0, v9);
const float v12 = indexArray(arrIn1_1, v9);
const float v13 = indexArray(arrIn1_0, v9);

x0 = fabsf(v12 - v10) > 1.0e-3f || fabsf(v13 - v11) > 1.0e-3f;
y0 = x0 || y0;
}
}
sdata0[threadIdx.x] = y0;
ix = min(shapeSize - blockIdx.x * blockDim.x, blockDim.x);
__syncthreads();
if (threadIdx.x + 256 < ix) {
x0 = sdata0[threadIdx.x + 256];
y0 = y0 || x0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 128 < ix) {
x0 = sdata0[threadIdx.x + 128];
y0 = y0 || x0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x + 64 < ix) {
x0 = sdata0[threadIdx.x + 64];
y0 = y0 || x0;
}
__syncthreads();
sdata0[threadIdx.x] = y0;
__syncthreads();
if (threadIdx.x < 32) {
if (threadIdx.x + 32 < ix) {
x0 = sdata0[threadIdx.x + 32];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 16 < ix) {
x0 = sdata0[threadIdx.x + 16];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 8 < ix) {
x0 = sdata0[threadIdx.x + 8];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 4 < ix) {
x0 = sdata0[threadIdx.x + 4];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 2 < ix) {
x0 = sdata0[threadIdx.x + 2];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
if (threadIdx.x + 1 < ix) {
x0 = sdata0[threadIdx.x + 1];
y0 = y0 || x0;
sdata0[threadIdx.x] = y0;
}
}
if (threadIdx.x == 0) {
if (shapeSize > 0) {
if (gridDim.x == 1) {
x0 = 0;
y0 = x0 || y0;
}
arrOut_0[blockIdx.x] = y0;
} else {
arrOut_0[blockIdx.x] = 0;
}
}
}
