#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrStencil_1;
static TexFloat arrStencil_0;
extern "C" __global__ void stencil(const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_0, const Int64 shStencil_1, const Int64 shStencil_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;

if (__all(sh1 >= 1 && sh1 < shOut_1 - 1 && (sh0 >= 1 && sh0 < shOut_0 - 1))) {
const Int64 w1 = (sh1 + -1) * shStencil_0 + (sh0 + 0);
const Int64 w3 = (sh1 + 0) * shStencil_0 + (sh0 + -1);
const Int64 w4 = (sh1 + 0) * shStencil_0 + (sh0 + 1);
const Int64 w6 = (sh1 + 1) * shStencil_0 + (sh0 + 0);
const float x14 = indexArray(arrStencil_0, w1);
const float x11 = indexArray(arrStencil_1, w3);
const float x7 = indexArray(arrStencil_1, w4);
const float x2 = indexArray(arrStencil_0, w6);

arrOut_0[ix] = -(0.5f * (x7 - x11 + x2 - x14));
} else {
const int w3 = ({ const Int64 _i = sh1 + -1;  _i >= 0 && _i < shStencil_1; }) && ({ const Int64 _i = sh0 + 0;  _i >= 0 && _i < shStencil_0; });
const Int64 w2 = (sh1 + -1) * shStencil_0 + (sh0 + 0);
const int w7 = ({ const Int64 _i = sh1 + 0;  _i >= 0 && _i < shStencil_1; }) && ({ const Int64 _i = sh0 + -1;  _i >= 0 && _i < shStencil_0; });
const Int64 w6 = (sh1 + 0) * shStencil_0 + (sh0 + -1);
const int w9 = ({ const Int64 _i = sh1 + 0;  _i >= 0 && _i < shStencil_1; }) && ({ const Int64 _i = sh0 + 1;  _i >= 0 && _i < shStencil_0; });
const Int64 w8 = (sh1 + 0) * shStencil_0 + (sh0 + 1);
const int w13 = ({ const Int64 _i = sh1 + 1;  _i >= 0 && _i < shStencil_1; }) && ({ const Int64 _i = sh0 + 0;  _i >= 0 && _i < shStencil_0; });
const Int64 w12 = (sh1 + 1) * shStencil_0 + (sh0 + 0);
const float x14 = w3 ? indexArray(arrStencil_0, w2) : 0.0f;
const float x11 = w7 ? indexArray(arrStencil_1, w6) : 0.0f;
const float x7 = w9 ? indexArray(arrStencil_1, w8) : 0.0f;
const float x2 = w13 ? indexArray(arrStencil_0, w12) : 0.0f;

arrOut_0[ix] = -(0.5f * (x7 - x11 + x2 - x14));
}
}
}
