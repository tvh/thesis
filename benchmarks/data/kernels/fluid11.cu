#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_1;
static TexFloat arrIn0_0;
static TexFloat arrIn1_0;
extern "C" __global__ void generate(const Int64 shIn0_1, const Int64 shIn0_0, const Int64 shIn1_1, const Int64 shIn1_0, const Int64 shOut_1, const Int64 shOut_0, float* __restrict__ arrOut_0)
{
const int shapeSize = shOut_1 * shOut_0;
const int gridSize = __umul24(blockDim.x, gridDim.x);
int ix;

for (ix = __umul24(blockDim.x, blockIdx.x) + threadIdx.x; ix < shapeSize; ix += gridSize) {
const Int64 tmp_0 = ix;
const Int64 tmp_1 = tmp_0 / shOut_0;
const Int64 sh1 = tmp_1 % shOut_1;
const Int64 sh0 = tmp_0 % shOut_0;
const Int64 v0 = (Int64) 1;
const Int64 v1 = (Int64) 0;
const Int64 v2 = sh1 * shIn0_0 + sh0;
const float v3 = indexArray(arrIn0_1, v2);
const float v4 = indexArray(arrIn0_0, v2);
const float v5 = (float) shIn0_0;
const float v6 = fmaxf(-0.5f, fminf(0.5f + v5, (float) sh0 - 0.1f * v5 * v3));
const float v7 = (float) shIn0_1;
const float v8 = fmaxf(-0.5f, fminf(0.5f + v7, (float) sh1 - 0.1f * v7 * v4));
const Int64 v9 = (Int64) -1 + (Int64) truncf(1.0f + v6);
const Int64 v10 = (Int64) -1 + (Int64) truncf(1.0f + v8);
const float v11 = v6 - (float) v9;
const Int64 v12 = v0 + v10;
const float v13 = v8 - (float) v10;
const float v14 = 1.0f - v13;
const Word8 v15 = v10 < v1 || (v9 < v1 || (v10 >= shIn0_1 || v9 >= shIn0_0));
const Int64 v16 = v10 * shIn1_0 + v9;
const Word8 v17 = v12 < v1 || (v9 < v1 || (v12 >= shIn0_1 || v9 >= shIn0_0));
const Int64 v18 = v12 * shIn1_0 + v9;
const Int64 v19 = v0 + v9;
const Word8 v20 = v10 < v1 || (v19 < v1 || (v10 >= shIn0_1 || v19 >= shIn0_0));
const Int64 v21 = v10 * shIn1_0 + v19;
const Word8 v22 = v12 < v1 || (v19 < v1 || (v12 >= shIn0_1 || v19 >= shIn0_0));
const Int64 v23 = v12 * shIn1_0 + v19;

arrOut_0[ix] = (1.0f - v11) * (v14 * (v15 ? 0.0f : indexArray(arrIn1_0, v16)) + v13 * (v17 ? 0.0f : indexArray(arrIn1_0, v18))) + v11 * (v14 * (v20 ? 0.0f : indexArray(arrIn1_0, v21)) + v13 * (v22 ? 0.0f : indexArray(arrIn1_0, v23)));
}
}
