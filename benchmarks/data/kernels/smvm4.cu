#include "hip/hip_runtime.h"
#include <accelerate_cuda.h>
static TexFloat arrIn0_0;
static TexFloat arrIn1_0;
static TexInt32 arrIn2_0;
static TexInt32 arrIn3_0;
extern "C" __global__ void foldSeg(const Int64 shIn0_0, const Int64 shIn1_0, const Int64 shIn2_0, const Int64 shIn3_0, const Int64 shOut_0, float* __restrict__ arrOut_0)
{
const int vectors_per_block = blockDim.x / warpSize;
const int num_vectors = __umul24(vectors_per_block, gridDim.x);
const int thread_id = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;
const int vector_id = thread_id / warpSize;
const int thread_lane = threadIdx.x & warpSize - 1;
const int vector_lane = threadIdx.x / warpSize;
const int num_segments = shOut_0;
const int total_segments = shOut_0;
int seg;
int ix;
extern volatile __shared__ int s_ptrs[][2];
volatile float* sdata0 = (float*) &s_ptrs[vectors_per_block][2];
float x0;
float y0;
const Int64 sh0 = min(shIn2_0, shIn1_0);

for (seg = vector_id; seg < total_segments; seg += num_vectors) {
const int s = seg % num_segments;
const int base = seg / num_segments * sh0;

if (thread_lane < 2) {
s_ptrs[vector_lane][thread_lane] = indexArray(arrIn3_0, s + thread_lane);
}

const int start = base + s_ptrs[vector_lane][0];
const int end = base + s_ptrs[vector_lane][1];
const int num_elements = end - start;

if (num_elements > warpSize) {
ix = start - (start & warpSize - 1) + thread_lane;
if (ix >= start) {
const Int64 v2 = ({ assert(ix >= 0 && ix < min(shIn2_0, shIn1_0)); ix; });
const Int64 v3 = (Int64) indexArray(arrIn2_0, v2);

y0 = indexArray(arrIn0_0, v3) * indexArray(arrIn1_0, v2);
}
if (ix + warpSize < end) {
const Int64 v2 = ({ assert(ix + warpSize >= 0 && ix + warpSize < min(shIn2_0, shIn1_0)); ix + warpSize; });
const Int64 v3 = (Int64) indexArray(arrIn2_0, v2);

x0 = indexArray(arrIn0_0, v3) * indexArray(arrIn1_0, v2);
if (ix >= start) {
y0 = x0 + y0;
} else {
y0 = x0;
}
}
for (ix += 2 * warpSize; ix < end; ix += warpSize) {
const Int64 v2 = ({ assert(ix >= 0 && ix < min(shIn2_0, shIn1_0)); ix; });
const Int64 v3 = (Int64) indexArray(arrIn2_0, v2);

x0 = indexArray(arrIn0_0, v3) * indexArray(arrIn1_0, v2);
y0 = x0 + y0;
}
} else if (start + thread_lane < end) {
const Int64 v2 = ({ assert(start + thread_lane >= 0 && start + thread_lane < min(shIn2_0, shIn1_0)); start + thread_lane; });
const Int64 v3 = (Int64) indexArray(arrIn2_0, v2);

y0 = indexArray(arrIn0_0, v3) * indexArray(arrIn1_0, v2);
}
ix = min(num_elements, warpSize);
sdata0[threadIdx.x] = y0;
if (thread_lane + 32 < ix) {
x0 = sdata0[threadIdx.x + 32];
y0 = y0 + x0;
sdata0[threadIdx.x] = y0;
}
if (thread_lane + 16 < ix) {
x0 = sdata0[threadIdx.x + 16];
y0 = y0 + x0;
sdata0[threadIdx.x] = y0;
}
if (thread_lane + 8 < ix) {
x0 = sdata0[threadIdx.x + 8];
y0 = y0 + x0;
sdata0[threadIdx.x] = y0;
}
if (thread_lane + 4 < ix) {
x0 = sdata0[threadIdx.x + 4];
y0 = y0 + x0;
sdata0[threadIdx.x] = y0;
}
if (thread_lane + 2 < ix) {
x0 = sdata0[threadIdx.x + 2];
y0 = y0 + x0;
sdata0[threadIdx.x] = y0;
}
if (thread_lane + 1 < ix) {
x0 = sdata0[threadIdx.x + 1];
y0 = y0 + x0;
sdata0[threadIdx.x] = y0;
}
if (thread_lane == 0) {
if (num_elements > 0) {
x0 = 0.0f;
y0 = x0 + y0;
} else {
y0 = 0.0f;
}
arrOut_0[seg] = y0;
}
}
}
