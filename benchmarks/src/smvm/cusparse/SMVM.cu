
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "matrix.h"
#include "Timing.h"

#define CUDA_CHECK(action)                                                      \
    do {                                                                        \
        if ( hipSuccess != action ) {                                          \
            printf("Failed: %s\n", #action);                                    \
            exit(1);                                                            \
        }                                                                       \
    } while (0)

#define CUSPARSE_CHECK(action)                                                  \
    do {                                                                        \
        if ( HIPSPARSE_STATUS_SUCCESS != action ) {                              \
            printf("Failed: %s\n", #action);                                    \
            exit(1);                                                            \
        }                                                                       \
    } while (0)

int main(int argc, char **argv)
{
    int num_rows;
    int num_cols;
    int num_entries;
    int *row_indices;
    int *col_indices;
    float *vals;

    if (argc < 2) {
        printf("usage: smvm <matrix market file>\n");
        exit(1);
    }

    // Read in matrix market file
    printf("Reading matrix...\n");
    read_matrix_market_file(argv[1], &num_rows, &num_cols, &num_entries, &row_indices, &col_indices, &vals);

    printf("Matrix %s: <%d, %d> with %d entries\n", argv[1], num_rows, num_cols, num_entries);

//    for (int i = 0; i < num_entries; ++i) {
//        printf("%8d%8d%16.2f\n", row_indices[i], col_indices[i], vals[i]);
//    }

    // Copy matrix data to GPU
    printf("Copy data to GPU...\n");
    int *d_row_indices;
    int *d_col_indices;
    float *d_vals;

    CUDA_CHECK( hipMalloc((void**) &d_row_indices, num_entries * sizeof(int)) );
    CUDA_CHECK( hipMalloc((void**) &d_col_indices, num_entries * sizeof(int)) );
    CUDA_CHECK( hipMalloc((void**) &d_vals,        num_entries * sizeof(int)) );

    CUDA_CHECK( hipMemcpy(d_row_indices, row_indices, num_entries * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(d_col_indices, col_indices, num_entries * sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(d_vals,        vals,        num_entries * sizeof(int), hipMemcpyHostToDevice) );

    // generate random dense vector
    printf("Generate dense vector...\n");
    float *x    = (float*) malloc(num_cols * sizeof(float));
    float *d_x;

    srand(1234);
    for (int i = 0; i < num_cols; ++i) {
        x[i]    = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }

    CUDA_CHECK( hipMalloc((void**) &d_x, num_cols * sizeof(float)) );
    CUDA_CHECK( hipMemcpy(d_x, x, num_cols * sizeof(float), hipMemcpyHostToDevice) );

    // Initialise CUSPARSE library and create matrix descriptor
    printf("Initialise CUDA sparse library...\n");
    hipsparseHandle_t handle     = NULL;
    hipsparseMatDescr_t descr    = NULL;

    CUSPARSE_CHECK( hipsparseCreate(&handle) );
    CUSPARSE_CHECK( hipsparseCreateMatDescr(&descr) );

    CUSPARSE_CHECK( hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL) );
    CUSPARSE_CHECK( hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO) );

    // Convert matrix from COO format to CSR
    printf("Convert COO to CSR format...\n");
    int *d_offsets;
    CUDA_CHECK( hipMalloc((void**) &d_offsets, (num_rows + 1) * sizeof(int)) );
    CUSPARSE_CHECK( hipsparseXcoo2csr(handle, d_row_indices, num_entries, num_rows, d_offsets, HIPSPARSE_INDEX_BASE_ZERO) );

    // Finally, do the matrix multiplication a bunch of times.
    printf("Running benchmark...\n");
    float *d_y;
    CUDA_CHECK( hipMalloc((void**) &d_y, num_rows * sizeof(float)) );

    struct benchtime* bt = bench_begin();
    float zero = 0.0;
    float one  = 1.0;

    for (int i = 0; i < 100; ++i) {
        CUSPARSE_CHECK( hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    num_rows, num_cols, num_entries,
                    &one, descr, d_vals,
                    d_offsets,
                    d_col_indices,
                    d_x,
                    &zero,
                    d_y) );
    }

    bench_end(bt);
    bench_done(bt);

    // Free memory
    free(row_indices);
    free(col_indices);
    free(vals);
    free(x);

    hipFree(d_row_indices);
    hipFree(d_col_indices);
    hipFree(d_vals);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_offsets);

    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(handle);

    return 0;
}
